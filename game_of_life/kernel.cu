#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <SDL.h>

using namespace std;

const int CELL_SIZE = 10; // Size of each cell in pixels

// Kernel function for Game of Life logic
__global__ void gameKernel(int* Md, int* Nd, int Width, int Height)
{
    int current = blockIdx.x * blockDim.x + threadIdx.x;
    if (current >= Width * Height) return; // Prevent out-of-bounds access

    int row = current / Width;
    int col = current % Width;

    int alive_neighbors = 0;

    // Check all 8 neighbors
    for (int i = -1; i <= 1; ++i) {
        for (int j = -1; j <= 1; ++j) {
            if (i == 0 && j == 0) continue; // Skip the cell itself
            int neighbor_row = row + i;
            int neighbor_col = col + j;
            if (neighbor_row >= 0 && neighbor_row < Height && neighbor_col >= 0 && neighbor_col < Width) {
                alive_neighbors += Md[neighbor_row * Width + neighbor_col];
            }
        }
    }

    // Apply Game of Life rules
    Nd[current] = (Md[current] == 1 && (alive_neighbors == 2 || alive_neighbors == 3)) || (Md[current] == 0 && alive_neighbors == 3) ? 1 : 0;
}

// Function to create a random board
int* createBoard(int width, int height) {
    int* board = new int[width * height];
    srand(static_cast<unsigned int>(time(nullptr)));
    for (int i = 0; i < width * height; i++) {
        board[i] = rand() % 2; // Randomly assign cells as 1 (alive) or 0 (dead)
    }
    return board;
}


// Function to render the board using SDL
void renderBoard(SDL_Renderer* renderer, int* board, int width, int height) {
    SDL_SetRenderDrawColor(renderer, 0, 200, 200, 255); // Bbackground
    SDL_RenderClear(renderer);

    SDL_SetRenderDrawColor(renderer, 0, 0, 0, 255); // Alive cells
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            if (board[y * width + x] == 1) {
                SDL_Rect cell = { x * CELL_SIZE, y * CELL_SIZE, CELL_SIZE, CELL_SIZE };
                SDL_RenderFillRect(renderer, &cell); // Draw the cell
            }
        }
    }

    SDL_RenderPresent(renderer); // Update the screen
}

// Function to ask for and validate the width
int askWidth(int maxThreads) {
    int width;
    cout << "\nEnter the width of the board: ";
    cin >> width;
    while (cin.fail() || (width <= 0) || (width > maxThreads)) {
        cout << "\nERROR: Incorrect width,  Enter the numerical value. ";
        cin.clear();
        cin.ignore(256, '\n');
        cout << "\nEnter the width of the board: ";
        cin >> width;
    }
    return width;
}

// Function to ask for and validate the height
int askHeight(int maxThreads) {
    int height;
    cout << "\nEnter the height of the board: ";
    cin >> height;
    while (cin.fail() || (height <= 0) || (height > maxThreads)) {
        cout << "\nERROR: Incorrect height, Enter the numerical value. ";
        cin.clear();
        cin.ignore(256, '\n');
        cout << "\nEnter the height of the board: ";
        cin >> height;
    }
    return height;
}

int main(int argc, char* args[]) {
    // Initialize CUDA
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    int threadsPerBlock = properties.maxThreadsPerBlock;

    // Ask user for board dimensions
    int width = askWidth(threadsPerBlock);
    int height = askHeight(threadsPerBlock);

    // Create and initialize the board
    int* board = createBoard(width, height);
    int* d_board, * d_nextBoard;
    hipMalloc(&d_board, width * height * sizeof(int));
    hipMalloc(&d_nextBoard, width * height * sizeof(int));
    hipMemcpy(d_board, board, width * height * sizeof(int), hipMemcpyHostToDevice);

    // Initialize SDL
    if (SDL_Init(SDL_INIT_VIDEO) < 0) {
        cout << "SDL could not initialize! SDL_Error: " << SDL_GetError() << endl;
        return -1;
    }

    SDL_Window* window = SDL_CreateWindow("Game of Life", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, width * CELL_SIZE, height * CELL_SIZE, SDL_WINDOW_SHOWN);
    SDL_Renderer* renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);

    bool quit = false;
    SDL_Event e;

    while (!quit) {
        while (SDL_PollEvent(&e) != 0) {
            if (e.type == SDL_QUIT) {
                quit = true;
            }
        }

        // Launch kernel to compute next generation
        int blocks = (width * height + threadsPerBlock - 1) / threadsPerBlock;
        gameKernel << <blocks, threadsPerBlock >> > (d_board, d_nextBoard, width, height);

        // Copy next state back to host
        hipMemcpy(board, d_nextBoard, width * height * sizeof(int), hipMemcpyDeviceToHost);

        // Render the board using SDL
        renderBoard(renderer, board, width, height);

        // Swap pointers for next iteration
        int* temp = d_board;
        d_board = d_nextBoard;
        d_nextBoard = temp;

        SDL_Delay(120); // Delay for visual effect (120 ms per frame)
    }

    // Free resources
    hipFree(d_board);
    hipFree(d_nextBoard);
    delete[] board;

    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}
